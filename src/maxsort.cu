/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
#include <stdio.h>
#include <stdlib.h>

#include <iostream>
#include <algorithm>

#include <hip/hip_runtime.h>


struct Task {
	uint id;
	float time;

	Task(uint id, float time) {
		this->id = id;
		this->time = time;
	}

	Task() {
		this->id = 0;
		this->time = 0;
	}

	bool operator() (Task i,Task j) { return (i.time > j.time); }
};

struct Machine {
	int id;
	float cost;

	Machine() {
		this->id = 0;
		this->cost = 0;
	}

	bool operator() (Machine i,Machine j) { return (i.cost < j.cost); }
};

void min_min(Task* tasks, float* completion_times, int* task_map, bool* task_scheduled, Machine* machines,
		int t, int m, int max_time) {

	uint count = 0;
	uint q = 0;

	while(count < t) {

		float current_time = 0;

		int j = machines[q].id;
		int i = 0;

		while(count < t && i < t) {
			int task_id = tasks[j * t + i].id;

			if (!task_scheduled[task_id]) {
				current_time = completion_times[j] + tasks[j * t + i].time;

				if(current_time > max_time){
					i++;
					continue;
				}

				task_scheduled[task_id] = true;
				task_map[task_id] = j;
				completion_times[j] = current_time;
				count++;
			}
			i++;
		}

		q++;

		if(q == m && count != t) {
			printf("### ERROR ###\n");
		}
	}
}

void machine_sorting(Machine* machines, int m) {

	std::stable_sort (&machines[0], &machines[0]+m, Machine());
}

void segmented_sorting(Task* tasks, int m, int t) {

	for(int i = 0; i < m; i++) {
		int j = i*t;
		std::stable_sort (&tasks[j], &tasks[j]+t, Task());
	}
}

template<typename T>
void print(T* vec, uint t, uint m) {
	std::cout << "\n";
	for (uint i = 0; i < t; i++) {
		for (uint j = 0; j < m; j++) {
			std::cout << vec[i * m + j] << " ";
		}
		std::cout << "\n";
	}

}

template<typename T>
void print(T* vec, uint t) {
	std::cout << "\n";
	for (uint i = 0; i < t; i++) {
		std::cout << vec[i] << " ";
	}
	std::cout << "\n";
}

void print(Task* vec, uint t, uint m) {
	std::cout << "\n";
	for (uint j = 0; j < m; j++) {
		for (uint i = 0; i < t; i++) {
			std::cout << "id=" << vec[j * t + i].id << " time="
					<< vec[j * t + i].time << "\t";
		}
		std::cout << "\n";
	}
}


void print(Machine* vec, uint m) {
	std::cout << "\n";
	for (uint j = 0; j < m; j++) {
			std::cout << "id=" << vec[j].id << " time="
					<< vec[j].cost << "\t";
	}
	std::cout << "\n";
}

void print(float* completion_times, Machine* vec, uint m) {
	float sum = 0;
	for (uint j = 0; j < m; j++) {
		uint id = vec[j].id;
		float cost = vec[j].cost * completion_times[id];

		std::cout << vec[j].cost << " * " << completion_times[id] << " = " << cost << "\n";
		sum += cost;
	}
	std::cout << "Custo Total: " << sum << "\n";
}

int main(int argc, char **argv) {
	int t, m;
	float max_time, aux;

	aux = scanf("%d", &t);
	aux = scanf("%d", &m);
	aux = scanf("%f", &max_time);

	//std::cout << "t=" << t << " m=" << m << "\n";

	Task *tasks = (Task *) malloc(sizeof(Task) * (t * m));
	bool *task_scheduled = (bool *) malloc(sizeof(bool) * t);
	int *task_map = (int *) malloc(sizeof(int) * (t));
	float *completion_times = (float *) malloc(sizeof(float) * (m));
	Machine *machines = (Machine *) malloc(sizeof(Machine) * (m));

	// Read matrix task machine
	for (int i = 0; i < t; i++) {
		for (int j = 0; j < m; j++) {
			int a = scanf("%f", &aux);
			tasks[j * t + i].id = i;
			tasks[j * t + i].time = aux;
			completion_times[j] = 0;
		}
		task_map[i] = -1;
		task_scheduled[i] = false;
	}

	//print(tasks, t, m);

	// Reading vector of costs for each machine
	for (int j = 0; j < m; j++) {
		int a = scanf("%f", &aux);
		machines[j].id = j;
		machines[j].cost = aux;
	}

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);
	machine_sorting(machines, m);
	//print(machines, m);

	segmented_sorting(tasks, m, t);
	//print(tasks,t,m);

	min_min(tasks, completion_times, task_map, task_scheduled, machines, t, m, max_time);
	hipEventRecord(stop);

	if (ELAPSED_TIME == 1) {
		hipEventSynchronize(stop);
		float milliseconds = 0;
		hipEventElapsedTime(&milliseconds, start, stop);
		std::cout << milliseconds << "\n";
	}
	else {
		//print(tasks, t, m);
		//print(completion_times, m);
		print(completion_times, machines, m);
		//print(task_scheduled, t);
		//print(task_map, t, m);
	}

	free(task_scheduled);
	free(task_map);
	free(tasks);
	free(completion_times);

	return 0;
}

